#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

#include <tvm/runtime/ndarray.h>
#include <tvm/runtime/packed_func.h>
#include <tvm/runtime/registry.h>

#include "msccl.cuh"

namespace tvm {
namespace runtime {

template <typename T>
hipError_t allreduce(const T* buff, T* scratch, T* resultBuff,
                      mscclpp::DeviceHandle<mscclpp::SmChannel>* smChannels,
                      mscclpp::DeviceHandle<mscclpp::SmChannel>* smOutChannels, int rank,
                      int nRanksPerNode, int worldSize, size_t nelems, hipStream_t stream);

MSCCL_API mscclResult_t mscclAllReduce(const void* sendbuff, void* recvbuff, size_t count,
                                       mscclDataType_t datatype, mscclRedOp_t op, mscclComm_t comm,
                                       hipStream_t stream) {
  size_t bytes = count * mscclTypeSize(datatype);
  if (sendbuff == nullptr || recvbuff == nullptr || bytes == 0 || comm == nullptr ||
      op != mscclSum || bytes > (1 << 24)) {
    return mscclInvalidArgument;
  }

  int rank = comm->comm->bootstrap()->getRank();
  channelKey key{sendbuff, recvbuff, bytes};
  mscclpp::DeviceHandle<mscclpp::SmChannel>* smChannels = nullptr;
  mscclpp::DeviceHandle<mscclpp::SmChannel>* smOutChannels = nullptr;

  auto it = comm->channelInfos.find(key);
  if (it == comm->channelInfos.end()) {
    // setup smChannels (src: sendbuff, dst: remote scratch buff)
    std::vector<mscclpp::SmChannel> channels =
        setupSmChannels(comm, comm->remoteScratchRegMemories, const_cast<void*>(sendbuff));
    ChannelInfo channelInfo{channels, {}, setupSmChannelDeviceHandles(channels), nullptr};
    it = comm->channelInfos.emplace(key, channelInfo).first;

    // TODO(csullivan): Consider supporting allreduce for larger transfers
    // setup smOutChannels (src: recvbuff, dst: remote recvbuff)
    // if (bytes > (1 << 24)) {
    //   std::vector<mscclpp::RegisteredMemory> remoteMemories =
    //       setupRemoteMemories(comm->comm, rank, recvbuff, bytes, mscclpp::Transport::CudaIpc);
    //   std::vector<mscclpp::SmChannel> outChannels = setupSmChannels(comm, remoteMemories,
    //   recvbuff); it->second.smOutChannels = outChannels; it->second.smOutChannelDeviceHandles =
    //   setupSmChannelDeviceHandles(outChannels);
    // }
  }

  smChannels = it->second.smChannelDeviceHandles.get();
  smOutChannels = it->second.smOutChannelDeviceHandles.get();

  switch (datatype) {
    case mscclFloat16:
      CUDACHECK(allreduce(reinterpret_cast<const half*>(sendbuff),
                          reinterpret_cast<half*>(comm->scratchBuff.get()),
                          reinterpret_cast<half*>(recvbuff), smChannels, smOutChannels, rank,
                          NRANKS_PER_NODE, comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    case mscclFloat32:
      CUDACHECK(allreduce(reinterpret_cast<const float*>(sendbuff),
                          reinterpret_cast<float*>(comm->scratchBuff.get()),
                          reinterpret_cast<float*>(recvbuff), smChannels, smOutChannels,
                          comm->comm->bootstrap()->getRank(), NRANKS_PER_NODE,
                          comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    case mscclInt32:
    case mscclUint32:
      CUDACHECK(allreduce(reinterpret_cast<const int*>(sendbuff),
                          reinterpret_cast<int*>(comm->scratchBuff.get()),
                          reinterpret_cast<int*>(recvbuff), smChannels, smOutChannels,
                          comm->comm->bootstrap()->getRank(), NRANKS_PER_NODE,
                          comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    default:
      return mscclInvalidArgument;
  }
  return mscclSuccess;
}

template <typename T>
__global__ void __launch_bounds__(1024, 1)
    allreduce_simple(mscclpp::SmChannelDeviceHandle* smChans, const T* buff, T* scratch,
                     void* resultBuff, int rank, int worldSize, size_t nelems,
                     const uint32_t flag) {
  nelems = nelems / (sizeof(int) / sizeof(T));

  const int nPeers = worldSize - 1;
  const size_t nPkts = nelems / 2;
  const int nelemsPerRank = nelems / worldSize;
  const int nPktsPerRank = nelemsPerRank / 2;
  const int nBlocksPerPeer = gridDim.x / nPeers;
  const int localBlockIdx = blockIdx.x % nBlocksPerPeer;
  const int peerIdx = blockIdx.x / nBlocksPerPeer;
  const int remoteRank = peerIdx < rank ? peerIdx : peerIdx + 1;
  mscclpp::SmChannelDeviceHandle smChan = smChans[peerIdx];
  const int tid = threadIdx.x + localBlockIdx * blockDim.x;

  size_t scratchOffset = rank * nPktsPerRank * sizeof(mscclpp::LLPacket);
  size_t resultOffset = 2 * nPkts * sizeof(mscclpp::LLPacket);
  size_t srcOffset = remoteRank * nelemsPerRank * sizeof(int);
  const uint2* src = reinterpret_cast<const uint2*>(reinterpret_cast<const char*>(buff) +
                                                    rank * nelemsPerRank * sizeof(int));
  uint2* dst = reinterpret_cast<uint2*>(reinterpret_cast<char*>(resultBuff) +
                                        rank * nelemsPerRank * sizeof(int));

  // Step 1. Write to scratch buffer which exposes memory to peers via cuda IPC memory
  smChan.putPackets(scratchOffset, srcOffset, nelemsPerRank * sizeof(int), tid,
                    blockDim.x * nBlocksPerPeer, flag);

  // Step 2. Get data from scratch buffer, reduce data, and write result back to peer scratch
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < nPktsPerRank;
       idx += blockDim.x * gridDim.x) {
    uint2 data = make_uint2(0, 0);
    for (int index = 0; index < nPeers; index++) {
      const int remoteRank = index < rank ? index : index + 1;
      mscclpp::LLPacket* dstPkt =
          reinterpret_cast<mscclpp::LLPacket*>(scratch) + remoteRank * nPktsPerRank;
      uint2 val = dstPkt[idx].read(flag);
      data = add_vectors<T>(val, data);
    }
    data = add_vectors<T>(data, src[idx]);
    dst[idx] = data;

    mscclpp::LLPacket packet;
    packet.data1 = data.x;
    packet.flag1 = flag;
    packet.data2 = data.y;
    packet.flag2 = flag;
    size_t offset = resultOffset / sizeof(mscclpp::LLPacket) + (idx + rank * nPktsPerRank);
    for (int index = 0; index < nPeers; index++) {
      smChans[index].write(offset, packet);
    }
  }

  // Step 3. Update local GPU's final result from peer scratch buffers
  mscclpp::LLPacket* dstPkt =
      reinterpret_cast<mscclpp::LLPacket*>(reinterpret_cast<char*>(scratch) + resultOffset);
  const int dstOffset = remoteRank * nPktsPerRank;
  uint2* result = reinterpret_cast<uint2*>(reinterpret_cast<char*>(resultBuff) +
                                           remoteRank * nelemsPerRank * sizeof(int));
  for (int idx = threadIdx.x + localBlockIdx * blockDim.x; idx < nPktsPerRank;
       idx += blockDim.x * nBlocksPerPeer) {
    uint2 data = dstPkt[idx + dstOffset].read(flag);
    result[idx].x = data.x;
    result[idx].y = data.y;
  }
}

template <typename T>
hipError_t allreduce(const T* buff, T* scratch, T* resultBuff,
                      mscclpp::DeviceHandle<mscclpp::SmChannel>* smChannels,
                      mscclpp::DeviceHandle<mscclpp::SmChannel>* smOutChannels, int rank,
                      int nRanksPerNode, int worldSize, size_t nelems, hipStream_t stream) {
  static uint32_t flag = 1;
  size_t num_bytes = sizeof(T) * nelems;
  ICHECK(num_bytes <= (1 << 24)) << "mscclpp allreduce expects bytes transfered < " << (1 << 24)
                                 << " but got num_bytes = " << num_bytes << " bytes";
  allreduce_simple<<<105, 1024, 0, stream>>>(smChannels, buff, scratch, resultBuff, rank, worldSize,
                                             nelems, flag++);

  return hipGetLastError();
}

}  // namespace runtime
}  // namespace tvm
