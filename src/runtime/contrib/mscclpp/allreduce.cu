#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

#include <tvm/runtime/ndarray.h>
#include <tvm/runtime/packed_func.h>
#include <tvm/runtime/registry.h>

#include "msccl.cuh"

namespace tvm {
namespace runtime {

template <typename T>
hipError_t allreduce(T* buff, T* scratch, T* resultBuff,
                      mscclpp::DeviceHandle<mscclpp::SmChannel>* smChannels,
                      mscclpp::DeviceHandle<mscclpp::SmChannel>* smOutChannels, int rank,
                      int nRanksPerNode, int worldSize, size_t nelems, hipStream_t stream);

MSCCL_API mscclResult_t mscclAllReduce(const void* sendbuff, void* recvbuff, size_t count,
                                       mscclDataType_t datatype, mscclRedOp_t op, mscclComm_t comm,
                                       hipStream_t stream) {
  size_t bytes = count * mscclTypeSize(datatype);
  if (sendbuff == nullptr || recvbuff == nullptr || bytes == 0 || comm == nullptr ||
      op != mscclSum || bytes > (1 << 24)) {
    return mscclInvalidArgument;
  }

  int rank = comm->comm->bootstrap()->getRank();
  channelKey key{sendbuff, recvbuff, bytes};
  mscclpp::DeviceHandle<mscclpp::SmChannel>* smChannels = nullptr;
  mscclpp::DeviceHandle<mscclpp::SmChannel>* smOutChannels = nullptr;

  auto it = comm->channelInfos.find(key);
  if (it == comm->channelInfos.end()) {
    // setup smChannels (src: sendbuff, dst: remote scratch buff)
    std::vector<mscclpp::SmChannel> channels =
        setupSmChannels(comm, comm->remoteScratchRegMemories, const_cast<void*>(sendbuff));
    ChannelInfo channelInfo{channels, {}, setupSmChannelDeviceHandles(channels), nullptr};
    it = comm->channelInfos.emplace(key, channelInfo).first;

    // TODO(csullivan): Consider supporting allreduce for larger transfers
    // setup smOutChannels (src: recvbuff, dst: remote recvbuff)
    // if (bytes > (1 << 24)) {
    //   std::vector<mscclpp::RegisteredMemory> remoteMemories =
    //       setupRemoteMemories(comm->comm, rank, recvbuff, bytes, mscclpp::Transport::CudaIpc);
    //   std::vector<mscclpp::SmChannel> outChannels = setupSmChannels(comm, remoteMemories,
    //   recvbuff); it->second.smOutChannels = outChannels; it->second.smOutChannelDeviceHandles =
    //   setupSmChannelDeviceHandles(outChannels);
    // }
  }

  smChannels = it->second.smChannelDeviceHandles.get();
  smOutChannels = it->second.smOutChannelDeviceHandles.get();

  switch (datatype) {
    case mscclFloat16:
      CUDACHECK(allreduce((half*)sendbuff, (half*)comm->scratchBuff.get(), (half*)recvbuff,
                          smChannels, smOutChannels, rank, NRANKS_PER_NODE,
                          comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    case mscclFloat32:
      CUDACHECK(allreduce((float*)sendbuff, (float*)comm->scratchBuff.get(), (float*)recvbuff,
                          smChannels, smOutChannels, comm->comm->bootstrap()->getRank(),
                          NRANKS_PER_NODE, comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    case mscclInt32:
    case mscclUint32:
      CUDACHECK(allreduce((int*)sendbuff, (int*)comm->scratchBuff.get(), (int*)recvbuff, smChannels,
                          smOutChannels, comm->comm->bootstrap()->getRank(), NRANKS_PER_NODE,
                          comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    default:
      return mscclInvalidArgument;
  }
  return mscclSuccess;
}

template <typename TYPE>
__global__ void __launch_bounds__(1024, 1)
    allreduce_simple(mscclpp::SmChannelDeviceHandle* smChans, TYPE* buff, TYPE* scratch,
                     void* resultBuff, int rank, int worldSize, size_t nelems,
                     const uint32_t flag) {
  nelems = nelems / (sizeof(int) / sizeof(TYPE));

  const int nPeers = worldSize - 1;
  const size_t nPkts = nelems / 2;
  const int nelemsPerRank = nelems / worldSize;
  const int nPktsPerRank = nelemsPerRank / 2;
  const int nBlocksPerPeer = gridDim.x / nPeers;
  const int localBlockIdx = blockIdx.x % nBlocksPerPeer;
  const int peerIdx = blockIdx.x / nBlocksPerPeer;
  const int remoteRank = peerIdx < rank ? peerIdx : peerIdx + 1;
  mscclpp::SmChannelDeviceHandle smChan = smChans[peerIdx];
  const int tid = threadIdx.x + localBlockIdx * blockDim.x;

  size_t scratchOffset = rank * nPktsPerRank * sizeof(mscclpp::LLPacket);
  size_t resultOffset = 2 * nPkts * sizeof(mscclpp::LLPacket);
  size_t srcOffset = remoteRank * nelemsPerRank * sizeof(int);
  uint2* src = (uint2*)((char*)buff + rank * nelemsPerRank * sizeof(int));
  uint2* dst = (uint2*)((char*)resultBuff + rank * nelemsPerRank * sizeof(int));

  // Step 1. Write to scratch buffer which exposes memory to peers via cuda IPC memory
  smChan.putPackets(scratchOffset, srcOffset, nelemsPerRank * sizeof(int), tid,
                    blockDim.x * nBlocksPerPeer, flag);

  // Step 2. Get data from scratch buffer, reduce data, and write result back to peer scratch
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < nPktsPerRank;
       idx += blockDim.x * gridDim.x) {
    uint2 data = make_uint2(0, 0);
    for (int index = 0; index < nPeers; index++) {
      const int remoteRank = index < rank ? index : index + 1;
      mscclpp::LLPacket* dstPkt = (mscclpp::LLPacket*)scratch + remoteRank * nPktsPerRank;
      uint2 val = dstPkt[idx].read(flag);
      data = add_vectors<TYPE>(val, data);
    }
    data = add_vectors<TYPE>(data, src[idx]);
    dst[idx] = data;

    mscclpp::LLPacket packet;
    packet.data1 = data.x;
    packet.flag1 = flag;
    packet.data2 = data.y;
    packet.flag2 = flag;
    size_t offset = resultOffset / sizeof(mscclpp::LLPacket) + (idx + rank * nPktsPerRank);
    for (int index = 0; index < nPeers; index++) {
      smChans[index].write(offset, packet);
    }
  }

  // Step 3. Update local GPU's final result from peer scratch buffers
  mscclpp::LLPacket* dstPkt = (mscclpp::LLPacket*)((char*)scratch + resultOffset);
  const int dstOffset = remoteRank * nPktsPerRank;
  uint2* result = (uint2*)((char*)resultBuff + remoteRank * nelemsPerRank * sizeof(int));
  for (int idx = threadIdx.x + localBlockIdx * blockDim.x; idx < nPktsPerRank;
       idx += blockDim.x * nBlocksPerPeer) {
    uint2 data = dstPkt[idx + dstOffset].read(flag);
    result[idx].x = data.x;
    result[idx].y = data.y;
  }
}

template <typename T>
hipError_t allreduce(T* buff, T* scratch, T* resultBuff,
                      mscclpp::DeviceHandle<mscclpp::SmChannel>* smChannels,
                      mscclpp::DeviceHandle<mscclpp::SmChannel>* smOutChannels, int rank,
                      int nRanksPerNode, int worldSize, size_t nelems, hipStream_t stream) {
  static uint32_t flag = 1;
  size_t num_bytes = sizeof(T) * nelems;
  ICHECK(num_bytes <= (1 << 24)) << "mscclpp allreduce expects bytes transfered < " << (1 << 24)
                                 << " but got num_bytes = " << num_bytes << " bytes";
  allreduce_simple<<<105, 1024, 0, stream>>>(smChannels, buff, scratch, resultBuff, rank, worldSize,
                                             nelems, flag++);

  return hipGetLastError();
}

}  // namespace runtime
}  // namespace tvm
