#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_fp16.h>
#include <dmlc/logging.h>
#include <stdint.h>

#include "custom_allreduce_kernels.h"

namespace tensorrt_llm {

static inline __device__ void st_flag_release(uint32_t& flag, uint32_t* flag_addr) {
#if __CUDA_ARCH__ >= 700
  asm volatile("st.global.release.sys.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#else
  __threadfence_system();
  asm volatile("st.global.volatile.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ void ld_flag_acquire(uint32_t& flag, uint32_t* flag_addr) {
#if __CUDA_ARCH__ >= 700
  asm volatile("ld.global.acquire.sys.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#else
  asm volatile("ld.global.volatile.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// Type Converter that packs data format to 128 bits data type
//
using PackedFloat = union {
  int4 packed;
  float unpacked[4];
};

using PackedHalf = union {
  int4 packed;
  half2 unpacked[4];
};

template <typename T>
struct PackedOn16Bytes {};

template <>
struct PackedOn16Bytes<float> {
  using Type = PackedFloat;
};

template <>
struct PackedOn16Bytes<half> {
  using Type = PackedHalf;
};

#ifdef ENABLE_BF16
using PackedBFloat16 = union {
  int4 packed;
  __hip_bfloat162 unpacked[4];
};

template <>
struct PackedOn16Bytes<__hip_bfloat16> {
  using Type = PackedBFloat16;
};
#endif

// add two 128b data
template <typename T>
inline __device__ int4 add128b(T& a, T& b) {
  T c;
  c.unpacked[0] = a.unpacked[0] + b.unpacked[0];
  c.unpacked[1] = a.unpacked[1] + b.unpacked[1];
  c.unpacked[2] = a.unpacked[2] + b.unpacked[2];
  c.unpacked[3] = a.unpacked[3] + b.unpacked[3];
  return c.packed;
}

__inline__ __device__ void multi_gpu_barrier(uint32_t** signals, const uint32_t flag,
                                             const size_t rank, const size_t world_size,
                                             int const tidx, int const bidx) {
  // At the end of the function, we now that has least block 0 from all others GPUs have reached
  // that point.
  uint32_t volatile* my_signals = signals[rank];
  if (tidx < world_size) {
    // The 1st block notifies the other ranks.
    if (bidx == 0) {
      signals[tidx][rank] = flag;
    }

    // Busy-wait until all ranks are ready.
    while (my_signals[tidx] != flag) {
    }
  }

  // Make sure we can move on...
  __syncthreads();
}

__global__ void multiGpuBarrierKernel(AllReduceParams params) {
  multi_gpu_barrier(params.peer_barrier_ptrs_out, params.barrier_flag, params.local_rank,
                    params.ranks_per_node, threadIdx.x, blockIdx.x);
}

template <typename T, int RANKS_PER_NODE>
static __global__ void oneShotAllReduceKernel(AllReduceParams params) {
  int const bidx = blockIdx.x;
  int const tidx = threadIdx.x;

  // The number of elements packed into one for comms
  static constexpr int NUM_ELTS = 16 / sizeof(T);

  // Packed data type for comms
  using PackedStruct = typename PackedOn16Bytes<T>::Type;

  multi_gpu_barrier(params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank,
                    RANKS_PER_NODE, tidx, bidx);

  // The source pointers. Distributed round-robin for the different warps.
  T const* src_d[RANKS_PER_NODE];
#pragma unroll
  for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
    int rank = (params.local_rank + ii) % RANKS_PER_NODE;
    src_d[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
  }

  // The location in the destination array (load 8 fp16 or load 4 fp32 using LDG.128).
  size_t offset = bidx * params.elts_per_block + tidx * NUM_ELTS;
  // The end of the segment computed by that block.
  size_t max_offset = min((bidx + 1) * params.elts_per_block, params.elts_per_rank);

  // Each block accumulates the values from the different GPUs on the same node.
  for (size_t iter_offset = offset; iter_offset < max_offset;
       iter_offset += blockDim.x * NUM_ELTS) {
    // Iterate over the different ranks/devices on the node to load the values.
    PackedStruct vals[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
      vals[ii].packed = *reinterpret_cast<int4 const*>(&src_d[ii][iter_offset]);
    }

    // Sum the values from the different ranks.
    PackedStruct sums;
    sums.packed = {0, 0, 0, 0};
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
      sums.packed = add128b(sums, vals[ii]);
    }

    // Store to the destination buffer.
    *reinterpret_cast<int4*>(&reinterpret_cast<T*>(params.local_output_buffer_ptr)[iter_offset]) =
        sums.packed;
  }
}

template <typename T, int RANKS_PER_NODE>
static __global__ void twoShotAllReduceKernel(AllReduceParams params) {
  // The block index.
  int const bidx = blockIdx.x;
  // The thread index with the block.
  int const tidx = threadIdx.x;

  // The number of elements packed into one for comms
  static constexpr int NUM_ELTS = 16 / sizeof(T);

  // Packed data type for comms
  using PackedType = typename PackedOn16Bytes<T>::Type;

  // The location in the destination array (load 8 fp16 or load 4 fp32 using LDG.128).
  const size_t block_offset = bidx * params.elts_per_block + tidx * NUM_ELTS;
  const size_t block_start = params.rank_offset + block_offset;
  // The end of the segment computed by that block.
  size_t max_offset =
      min(block_start + params.elts_per_block, params.rank_offset + params.elts_per_rank);

  multi_gpu_barrier(params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank,
                    RANKS_PER_NODE, tidx, bidx);

  // The source pointers. Distributed round-robin for the different warps.
  T* src_d[RANKS_PER_NODE];
  // The destination ranks for round-robin gathering
  size_t dst_rank[RANKS_PER_NODE];
#pragma unroll
  for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
    int rank = (params.local_rank + ii) % RANKS_PER_NODE;
    src_d[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    dst_rank[ii] = rank;
  }

  // Each block accumulates the values from the different GPUs on the same node.
  for (size_t local_offset = block_start; local_offset < max_offset;
       local_offset += blockDim.x * NUM_ELTS) {
    // Iterate over the different ranks/devices on the node to load the values.
    PackedType vals[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
      vals[ii].packed = *reinterpret_cast<int4 const*>(&src_d[ii][local_offset]);
    }

    // Sum the values from the different ranks.
    PackedType sums;
    sums.packed = {0, 0, 0, 0};
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
      sums.packed = add128b(sums, vals[ii]);
    }

    // Store to the local buffer.
    *reinterpret_cast<int4*>(&src_d[0][local_offset]) = sums.packed;
  }

  // sync threads to make sure all block threads have the sums
  __syncthreads();

  // barriers among the blocks with the same idx (release-acquire semantics)
  if (tidx < RANKS_PER_NODE) {
    // The all blocks notifies the other ranks.
    uint32_t flag_block_offset = RANKS_PER_NODE + bidx * RANKS_PER_NODE;
    st_flag_release(params.barrier_flag,
                    params.peer_barrier_ptrs_in[tidx] + flag_block_offset + params.local_rank);

    // Busy-wait until all ranks are ready.
    uint32_t rank_barrier = 0;
    uint32_t* peer_barrier_d =
        params.peer_barrier_ptrs_in[params.local_rank] + flag_block_offset + tidx;
    do {
      ld_flag_acquire(rank_barrier, peer_barrier_d);
    } while (rank_barrier != params.barrier_flag);
  }

  // sync threads to make sure all other ranks has the final partial results
  __syncthreads();

  size_t max_block_offset = min(block_offset + params.elts_per_block, params.elts_per_rank);
  // Gather all needed elts from other intra-node ranks
  for (size_t local_offset = block_offset; local_offset < max_block_offset;
       local_offset += blockDim.x * NUM_ELTS) {
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
      // use round-robin gathering from other ranks
      size_t offset_rank = dst_rank[ii] * params.elts_per_rank + local_offset;
      if (offset_rank >= params.elts_total) {
        continue;
      }
      *reinterpret_cast<int4*>(&reinterpret_cast<T*>(params.local_output_buffer_ptr)[offset_rank]) =
          *reinterpret_cast<int4*>(&src_d[ii][offset_rank]);
    }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

inline int divUp(int a, int b) { return (a + b - 1) / b; }

std::tuple<int, int> kernelLaunchConfig(AllReduceStrategyType algo, AllReduceParams& param,
                                        size_t elts_per_thread) {
  ICHECK(param.elts_total % elts_per_thread == 0);

  int blocks_per_grid = 1, threads_per_block = DEFAULT_BLOCK_SIZE;

  const size_t total_threads = param.elts_total / elts_per_thread;
  switch (algo) {
    case AllReduceStrategyType::ONESHOT: {        // one stage all reduce algo
      if (total_threads <= DEFAULT_BLOCK_SIZE) {  // local reduce
        threads_per_block = WARP_SIZE * divUp(total_threads, WARP_SIZE);
        blocks_per_grid = 1;
      } else {  // local reduce
        threads_per_block = DEFAULT_BLOCK_SIZE;
        blocks_per_grid = divUp(total_threads, DEFAULT_BLOCK_SIZE);
        blocks_per_grid = std::min(static_cast<int>(MAX_ALL_REDUCE_BLOCKS), blocks_per_grid);
      }
      param.elts_per_rank = param.elts_total;
      param.elts_per_block =
          elts_per_thread * divUp(param.elts_per_rank, elts_per_thread * blocks_per_grid);
      break;
    }
    case AllReduceStrategyType::TWOSHOT: {  // two stage all reduce algo
      const size_t elts_per_rank = param.elts_total / param.ranks_per_node;
      ICHECK(elts_per_rank % elts_per_thread == 0);

      size_t total_threads = elts_per_rank / elts_per_thread;
      total_threads = WARP_SIZE * ((total_threads + WARP_SIZE - 1) / WARP_SIZE);
      ICHECK(total_threads % WARP_SIZE == 0);

      while (total_threads % blocks_per_grid != 0 ||
             total_threads / blocks_per_grid > DEFAULT_BLOCK_SIZE) {
        blocks_per_grid += 1;
      }

      threads_per_block = total_threads / blocks_per_grid;

      // NOTE: need to adjust here
      if (static_cast<size_t>(blocks_per_grid) > MAX_ALL_REDUCE_BLOCKS) {
        size_t iter_factor = 1;
        while (blocks_per_grid / iter_factor > MAX_ALL_REDUCE_BLOCKS ||
               blocks_per_grid % iter_factor) {
          iter_factor += 1;
        }
        blocks_per_grid /= iter_factor;
      }
      param.elts_per_rank = param.elts_total / param.ranks_per_node;
      param.elts_per_block = param.elts_per_rank / blocks_per_grid;
      param.elts_per_block = elts_per_thread * divUp(param.elts_per_block, elts_per_thread);
      param.rank_offset = param.rank * param.elts_per_rank;
      break;
    }
    default:
      LOG(FATAL) << ("Algorithm not supported here.");
  }

  return std::make_tuple(blocks_per_grid, threads_per_block);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T, int RANKS_PER_NODE>
void dispatchARKernels(AllReduceStrategyType algo, AllReduceParams& param, int blocks_per_grid,
                       int threads_per_block, hipStream_t stream) {
  if (algo == AllReduceStrategyType::ONESHOT) {
    oneShotAllReduceKernel<T, RANKS_PER_NODE>
        <<<blocks_per_grid, threads_per_block, 0, stream>>>(param);
  } else {
    twoShotAllReduceKernel<T, RANKS_PER_NODE>
        <<<blocks_per_grid, threads_per_block, 0, stream>>>(param);
  }
}

template <typename T>
void invokeOneOrTwoShotAllReduceKernel(AllReduceParams& param, AllReduceStrategyType strat,
                                       hipStream_t stream) {
  ICHECK(strat == AllReduceStrategyType::ONESHOT || strat == AllReduceStrategyType::TWOSHOT);
  auto last_error = hipGetLastError();
  if (last_error != hipSuccess) {
    LOG(INFO) << "cuda error:" << hipGetErrorString(last_error);
  }

  size_t elts_per_thread = 16 / sizeof(T);
  auto [blocks_per_grid, threads_per_block] = kernelLaunchConfig(strat, param, elts_per_thread);
  switch (param.ranks_per_node) {
    case 2:
      dispatchARKernels<T, 2>(strat, param, blocks_per_grid, threads_per_block, stream);
      break;
    case 4:
      dispatchARKernels<T, 4>(strat, param, blocks_per_grid, threads_per_block, stream);
      break;
    case 6:
      dispatchARKernels<T, 6>(strat, param, blocks_per_grid, threads_per_block, stream);
      break;
    case 8:
      dispatchARKernels<T, 8>(strat, param, blocks_per_grid, threads_per_block, stream);
      break;
    default:
      break;
  }
  last_error = hipGetLastError();
  if (last_error != hipSuccess) {
    LOG(INFO) << "cuda error:" << hipGetErrorString(last_error);
  }
}

void invokeMultiGpuBarrier(AllReduceParams& param, hipStream_t stream) {
  multiGpuBarrierKernel<<<1, param.ranks_per_node, 0, stream>>>(param);
}

void customAllReduce(AllReduceParams& params, void* data, size_t elts, DLDataType dataType,
                     AllReduceStrategyType strat, hipStream_t stream) {
  params.local_output_buffer_ptr = data;
  params.elts_total = elts;

  if (dataType.code == kDLFloat && dataType.bits == 32) {
    invokeOneOrTwoShotAllReduceKernel<float>(params, strat, stream);
  } else if (dataType.code == kDLFloat && dataType.bits == 16) {
    invokeOneOrTwoShotAllReduceKernel<half>(params, strat, stream);
  }
#ifdef ENABLE_BF16
  else if (dataType.code == kDLBfloat && dataType.bits == 16) {
    invokeOneOrTwoShotAllReduceKernel<__hip_bfloat16>(params, strat, stream);
  }
#endif
  else {
    LOG(FATAL) << ("Unsupported dataType for customAllReduce");
  }
}

}  // namespace tensorrt_llm
